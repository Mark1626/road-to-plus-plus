#include "hip/hip_runtime.h"
/*
  nvcc -o exp-3 exp-3.cu -O3 -Xcompiler -fopenmp -DASSERT
  ./exp2 2000 25

  This algorithm is not optimized for SIMT architecture, and runs into memory
  issues for larger window sizes

  Image size 100 100
  Window size 5 5
  Time taken CPU Grid: 4 ms
  threads: (16 16 1) blocks: (16 16 1)
  Elapsed GPU time 29.144129 ms
  threads: (16 16 1) blocks: (16 16 1)
  Elapsed GPU time Managed Memory 38.452671 ms
  Assertions complete
  Assertions complete managed
*/
#include <assert.h>
#include <chrono>
#include <cstdlib>
#include <stdio.h>

using namespace std::chrono;

const int NDIM = 2;
const int TOLERANCE = 0.001;

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "Cuda Runtime Error : %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

#define SWAP(a, b)                                                             \
  {                                                                            \
    float t = (a);                                                             \
    (a) = (b);                                                                 \
    (b) = t;                                                                   \
  }

__device__ __host__ float qselect(float *arr, int len, int nth) {
  int start = 0;
  for (int index = 0; index < len - 1; index++) {
    if (arr[index] > arr[len - 1])
      continue;
    SWAP(arr[index], arr[start]);
    start++;
  }
  SWAP(arr[len - 1], arr[start]);

  if (nth == start)
    return arr[start];

  return start > nth ? qselect(arr, start, nth)
                     : qselect(arr + start, len - start, nth - start);
}

void sliding_window(const float *arr, const bool *mask, float *res1,
                    float *res2, int shape[NDIM], int hboxsz[NDIM],
                    int resShape[NDIM]) {

  const int bxlen = hboxsz[0];
  const int bylen = hboxsz[1];
  const int xlen = shape[0];

#pragma omp parallel for collapse(2)
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      int pos[NDIM];
      int blc[NDIM];
      int trc[NDIM];

      pos[0] = x;
      pos[1] = y;

      blc[0] = pos[0];
      blc[1] = pos[1];

      trc[0] = blc[0] + hboxsz[0];
      trc[1] = blc[1] + hboxsz[1];

      float *tmp = new float[bxlen * bylen];
      int len = 0;

      for (int yy = blc[1]; yy < trc[1]; yy++) {
        for (int xx = blc[0]; xx < trc[0]; xx++) {
          int idx = yy * xlen + xx;
          if (mask[idx]) {
            tmp[len] = arr[idx];
            len++;
          }
        }
      }
      unsigned long nth = len / 2;

      float mid = qselect(tmp, len, nth);

      if (len % 2 == 0) {
        mid += qselect(tmp, len, nth - 1);
        mid /= 2.0;
      }

      // float mid = median(tmp, len);

      res1[y * resShape[0] + x] = mid;
      res2[y * resShape[0] + x] = mid;

      delete[] tmp;
    }
  }
}

__global__ void
gliding_window(const float *arr, const bool *mask, float *res1, float *res2,
               const unsigned int xlen, const unsigned int ylen,
               const unsigned int xlimit, const unsigned int ylimit,
               const unsigned int bxlen, const unsigned int bylen) {

  float *tmp = (float *)malloc(sizeof(float) * bxlen * bylen);
  // float tmp[bxlen * bylen];
  unsigned int ystride = blockDim.y * gridDim.y;
  unsigned int xstride = blockDim.x * gridDim.x;
  // printf("xstride: %d ystride: %d\n", xstride, ystride);

  for (unsigned int y = blockIdx.y * blockDim.y + threadIdx.y; y < ylimit;
       y += ystride) {
    for (unsigned int x = blockIdx.x * blockDim.x + threadIdx.x; x < xlimit;
         x += xstride) {
      // printf("x: %d y: %d\n", x, y);
      int pos[NDIM];
      int blc[NDIM];
      int trc[NDIM];

      pos[0] = x;
      pos[1] = y;

      blc[0] = pos[0];
      blc[1] = pos[1];

      trc[0] = blc[0] + bxlen;
      trc[1] = blc[1] + bylen;

      // float mid = gliding_median(arr, blc, trc, xlen, ylen, bxlen, bylen);
      int len = 0;

      if (tmp == NULL) {
        printf("Memory allocation failure %d %d\n", x, y);
        return;
      }

      // float sum = 0.0;
      // #pragma unroll
      for (int yy = blc[1]; yy < trc[1]; yy++) {
        for (int xx = blc[0]; xx < trc[0]; xx++) {
          int idx = yy * xlen + xx;
          if (mask[idx]) {
            tmp[len] = arr[idx];
            len++;
          }
        }
      }
      unsigned long nth = len / 2;

      // float mid = median(tmp, len);
      float mid = qselect(tmp, len, nth);

      if (len % 2 == 0) {
        mid += qselect(tmp, len, nth - 1);
        mid /= 2.0;
      }

      res1[y * xlimit + x] = mid;
      res2[y * xlimit + x] = mid;
    }
  }
  free(tmp);
}

#ifndef BLOCKSIZEX
#define BLOCKSIZEX 16
#endif

#ifndef GRIDSIZEX
#define GRIDSIZEX 16
#endif

#ifndef BLOCKSIZEY
#define BLOCKSIZEY 16
#endif

#ifndef GRIDSIZEY
#define GRIDSIZEY 16
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE 16
#endif

#ifndef GRIDSIZE
#define GRIDSIZE 16
#endif

void gpu_sliding_window(const float *arr, const bool *mask, float *res1,
                        float *res2, int shape[NDIM], int hboxsz[NDIM],
                        int resShape[NDIM]) {

  size_t arrSize = sizeof(float) * shape[0] * shape[1];
  size_t maskSize = sizeof(bool) * shape[0] * shape[1];
  size_t resSize = sizeof(float) * resShape[0] * resShape[1];

#ifndef GDEBUG
  dim3 dimBlock(BLOCKSIZEX, BLOCKSIZEY, 1);
#else
  dim3 dimBlock(1, 1, 1);
#endif

  int ylimit = resShape[1];
  int xlimit = resShape[0];

#ifndef GDEBUG
  int gridx = (xlimit + dimBlock.x - 1) / dimBlock.x;
  int gridy = (ylimit + dimBlock.y - 1) / dimBlock.y;
  // dim3 dimGrid(gridx, gridy, 1);
  dim3 dimGrid(GRIDSIZEX, GRIDSIZEY, 1);
#else
  dim3 dimGrid(1, 1, 1);
#endif

  printf("threads: (%d %d %d) blocks: (%d %d %d)\n", dimBlock.x, dimBlock.y,
         dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

  float *d_arr;
  bool *d_mask;
  float *d_res_1;
  float *d_res_2;

  hipEvent_t start, stop;

  checkCuda(hipEventCreate(&start));
  checkCuda(hipEventCreate(&stop));
  // printf("Starting\n");

  checkCuda(hipEventRecord(start));
  // printf("Starting 2\n");

  checkCuda(hipMalloc(&d_mask, maskSize));
  checkCuda(hipMalloc(&d_res_1, resSize));
  checkCuda(hipMalloc(&d_arr, arrSize));
  checkCuda(hipMalloc(&d_res_2, resSize));

  checkCuda(hipMemcpyAsync(d_mask, mask, maskSize, hipMemcpyHostToDevice));
  checkCuda(hipMemcpyAsync(d_arr, arr, arrSize, hipMemcpyHostToDevice));

  // Run the kernel
  gliding_window<<<dimGrid, dimBlock>>>(d_arr, d_mask, d_res_1, d_res_2,
                                        shape[0], shape[1], resShape[0],
                                        resShape[1], hboxsz[0], hboxsz[1]);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("Error execting kernel %s\n", hipGetErrorString(error));
    assert(error == hipSuccess);
  }

  // printf("Finished kernel\n");

  hipDeviceSynchronize();

  // printf("Synched\n");

  checkCuda(hipMemcpyAsync(res1, d_res_1, resSize, hipMemcpyDeviceToHost));
  checkCuda(hipMemcpyAsync(res2, d_res_2, resSize, hipMemcpyDeviceToHost));

  // printf("Copy back\n");

  checkCuda(hipEventRecord(stop));
  checkCuda(hipEventSynchronize(stop));

  float elapsedTime;
  checkCuda(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("Elapsed GPU time %f ms\n", elapsedTime);

  checkCuda(hipFree(d_arr));
  checkCuda(hipFree(d_res_1));
  checkCuda(hipFree(d_mask));
  checkCuda(hipFree(d_res_2));

  checkCuda(hipEventDestroy(start));
  checkCuda(hipEventDestroy(stop));
}

void gpu_sliding_window_managed(const float *arr, const bool *mask, float *res1,
                                float *res2, int shape[NDIM], int hboxsz[NDIM],
                                int resShape[NDIM]) {

  size_t arrSize = sizeof(float) * shape[0] * shape[1];
  size_t maskSize = sizeof(bool) * shape[0] * shape[1];
  size_t resSize = sizeof(float) * resShape[0] * resShape[1];
  size_t arrN = shape[0] * shape[1];

#ifndef GDEBUG
  dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);
#else
  dim3 dimBlock(1, 1, 1);
#endif

  int ylimit = resShape[1];
  int xlimit = resShape[0];

#ifndef GDEBUG
  int gridx = (xlimit + dimBlock.x - 1) / dimBlock.x;
  int gridy = (ylimit + dimBlock.y - 1) / dimBlock.y;
  // dim3 dimGrid(gridx, gridy, 1);
  dim3 dimGrid(GRIDSIZEX, GRIDSIZEY, 1);
#else
  dim3 dimGrid(1, 1, 1);
#endif

  printf("threads: (%d %d %d) blocks: (%d %d %d)\n", dimBlock.x, dimBlock.y,
         dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

  float *d_arr;
  float *d_res_1;
  bool *d_mask;
  float *d_res_2;

  int device = -1;
  checkCuda(hipGetDevice(&device));

  hipEvent_t start, stop;

  checkCuda(hipEventCreate(&start));
  checkCuda(hipEventCreate(&stop));
  // printf("Starting\n");

  checkCuda(hipEventRecord(start));
  // printf("Starting 2\n");

  checkCuda(hipMallocManaged(&d_arr, arrSize));
  checkCuda(hipMallocManaged(&d_mask, maskSize));
  checkCuda(hipMallocManaged(&d_res_1, resSize));
  checkCuda(hipMallocManaged(&d_res_2, resSize));

  // checkCuda(hipMemcpyAsync(d_arr, arr, arrSize, hipMemcpyHostToDevice));
  memcpy(d_arr, arr, arrSize);
  memcpy(d_mask, mask, maskSize);
  checkCuda(hipMemPrefetchAsync(d_arr, (arrN) * sizeof(float), device, NULL));
  checkCuda(hipMemPrefetchAsync(d_mask, (arrN) * sizeof(bool), device, NULL));

  // Run the kernel
  gliding_window<<<dimGrid, dimBlock>>>(d_arr, d_mask, d_res_1, d_res_2,
                                        shape[0], shape[1], resShape[0],
                                        resShape[1], hboxsz[0], hboxsz[1]);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("Error execting kernel %s\n", hipGetErrorString(error));
    assert(error == hipSuccess);
  }

  // printf("Finished kernel\n");

  hipDeviceSynchronize();

  // printf("Synched\n");

  memcpy(res1, d_res_1, resSize);
  memcpy(res2, d_res_2, resSize);
  // checkCuda(hipMemcpyAsync(res, d_res, resSize, hipMemcpyDeviceToHost));

  // printf("Copy back\n");

  checkCuda(hipEventRecord(stop));
  checkCuda(hipEventSynchronize(stop));

  float elapsedTime;
  checkCuda(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("Elapsed GPU time Managed Memory %f ms\n", elapsedTime);

  checkCuda(hipFree(d_arr));
  checkCuda(hipFree(d_res_1));
  checkCuda(hipFree(d_mask));
  checkCuda(hipFree(d_res_2));

  checkCuda(hipEventDestroy(start));
  checkCuda(hipEventDestroy(stop));
}

void experiment(int SIZE, int WINDOW_SIZE) {
  int shape[NDIM] = {SIZE, SIZE};
  int hboxsz[NDIM] = {WINDOW_SIZE, WINDOW_SIZE};
  int resShape[NDIM] = {shape[0] - hboxsz[0], shape[1] - hboxsz[1]};

  size_t arrSize = sizeof(float) * shape[0] * shape[1];
  size_t maskSize = sizeof(bool) * shape[0] * shape[1];
  size_t resSize = sizeof(float) * resShape[0] * resShape[1];

  size_t ndim = 2;

  float *arr = (float *)malloc(arrSize);
  bool *mask = (bool *)malloc(maskSize);

  double a = 5.0;

  for (int y = 0; y < shape[1]; y++) {
    for (int x = 0; x < shape[0]; x++) {
      float val = (double)std::rand() / (double)(RAND_MAX / a);
      arr[y * shape[0] + x] = val;
      mask[y * shape[0] + x] = val > 1.0;
    }
  }

#ifdef DEBUG
  printf("Data\n");
  printf("------------------------------\n");
  printf("\nArray\n");
  for (int y = 0; y < shape[1]; y++) {
    for (int x = 0; x < shape[0]; x++) {
      float val = arr[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }

  printf("\nMask\n");
  for (int y = 0; y < shape[1]; y++) {
    for (int x = 0; x < shape[0]; x++) {
      bool val = mask[y * resShape[0] + x];
      printf("%d ", val);
    }
    printf("\n");
  }

#endif

  float *res_cpu_1 = (float *)malloc(sizeof(float) * resShape[0] * resShape[1]);
  float *res_gpu_m_1 =
      (float *)malloc(sizeof(float) * resShape[0] * resShape[1]);
  float *res_gpu_1 = (float *)malloc(sizeof(float) * resShape[0] * resShape[1]);

  float *res_cpu_2 = (float *)malloc(sizeof(float) * resShape[0] * resShape[1]);
  float *res_gpu_m_2 =
      (float *)malloc(sizeof(float) * resShape[0] * resShape[1]);
  float *res_gpu_2 = (float *)malloc(sizeof(float) * resShape[0] * resShape[1]);

  printf("Image size %d %d\n", SIZE, SIZE);
  printf("Window size %d %d\n", WINDOW_SIZE, WINDOW_SIZE);

  auto t1 = high_resolution_clock::now();

  sliding_window(arr, mask, res_cpu_1, res_cpu_2, shape, hboxsz, resShape);

  auto t2 = high_resolution_clock::now();
  printf("Time taken CPU Grid: %ld ms\n",
         duration_cast<milliseconds>(t2 - t1).count());

  // hipDeviceSetLimit(hipLimitMallocHeapSize, 32 * 1024 * 1024);

  gpu_sliding_window(arr, mask, res_gpu_1, res_gpu_2, shape, hboxsz, resShape);

  gpu_sliding_window_managed(arr, mask, res_gpu_m_1, res_gpu_m_2, shape, hboxsz,
                             resShape);

#ifdef DEBUG
  printf("Result 1\n");
  printf("------------------------------\n");
  printf("\nExpected\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float val = res_cpu_1[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }

  printf("\nActual\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float val = res_gpu_1[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }

  printf("\nActual Managed\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float val = res_gpu_m_1[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }
  printf("\nResult 2\n");
  printf("------------------------------\n");
  printf("\nExpected\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float val = res_cpu_2[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }

  printf("\nActual\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float val = res_gpu_2[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }

  printf("\nActual Managed\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float val = res_gpu_m_2[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }
#endif

#ifdef ASSERT
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float actual = res_gpu_1[y * resShape[0] + x];
      float expected = res_cpu_1[y * resShape[0] + x];
      if (fabs(actual - expected) > TOLERANCE) {
        fprintf(stderr,
                "Assertion failed value 1 at %d %d expected: %f actual: %f\n",
                x, y, expected, actual);
      }

      actual = res_gpu_2[y * resShape[0] + x];
      expected = res_cpu_2[y * resShape[0] + x];
      if (fabs(actual - expected) > TOLERANCE) {
        fprintf(stderr,
                "Assertion failed value 2 at %d %d expected: %f actual: %f\n",
                x, y, expected, actual);
      }
    }
  }
  printf("Assertions complete\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float actual = res_gpu_m_1[y * resShape[0] + x];
      float expected = res_cpu_1[y * resShape[0] + x];
      if (fabs(actual - expected) > TOLERANCE) {
        fprintf(stderr,
                "Assertion failed value 1 at %d %d expected: %f actual: %f\n",
                x, y, expected, actual);
      }

      actual = res_gpu_m_2[y * resShape[0] + x];
      expected = res_cpu_2[y * resShape[0] + x];
      if (fabs(actual - expected) > TOLERANCE) {
        fprintf(stderr,
                "Assertion failed value 2 at %d %d expected: %f actual: %f\n",
                x, y, expected, actual);
      }
    }
  }
  printf("Assertions complete managed\n");
#endif

  free(res_gpu_m_1);
  free(res_gpu_1);
  free(res_cpu_1);
  free(res_gpu_m_2);
  free(res_gpu_2);
  free(res_cpu_2);
  free(arr);
  free(mask);
}

int main(int argc, char **argv) {
  if (argc < 3) {
    fprintf(stderr, "Usage:\n exp-2 <SIZE> <WINDOW_SIZE>\n");
    return 1;
  }
  int SIZE = std::atoi(argv[1]);
  int WINDOW_SIZE = std::atoi(argv[2]);
  experiment(SIZE, WINDOW_SIZE);
}
