/*
  Does not work for odd window sizes
*/

#include <hip/hip_runtime.h>
#include <assert.h>
#include <chrono>
#include <stdio.h>

using namespace std::chrono;

#define NDIM 2

const int TOLERANCE = 0.001;

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "Cuda Runtime Error : %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

#define SWAP(a, b)                                                             \
  {                                                                            \
    float t = (a);                                                             \
    (a) = (b);                                                                 \
    (b) = t;                                                                   \
  }

#define BLOCKX 16
#define BLOCKY 16
#define GRIDX 16
#define GRIDY 16

#define BXLEN 2
#define BYLEN 2

#define XBUFFER (BLOCKX + BXLEN - 1)
#define YBUFFER (BLOCKY + BYLEN - 1)

// #define DEBUG
// #define ASSERT

__global__ void
shared_mem_kernel(const float *in, float *out, const unsigned int xlen,
                  const unsigned int ylen, const unsigned int xlimit,
                  const unsigned int ylimit) {

  // float tmp[BXLEN][BYLEN];
  __shared__ float buffer[XBUFFER][YBUFFER];

  // printf("Buffer dim: %d, %d\n", bufferx, buffery);

  unsigned int ystride = blockDim.y * gridDim.y;
  unsigned int xstride = blockDim.x * gridDim.x;

  #ifdef DEBUG
  printf("xstride: %d ystride: %d\n", xstride, ystride);
  #endif
  // printf("x: %d y: %d\n", blockIdx.x * blockDim.x + threadIdx.x,
  //        blockIdx.y * blockDim.y + threadIdx.y);
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int tidy = blockIdx.y * blockDim.y + threadIdx.y;

#ifdef DEBUG
      printf("Array\n");
      for (int yy = 0; yy < ylen; yy++) {
        for (int xx = 0; xx < xlen; xx++) {
          printf("%f ", in[yy * xlen + xx]);
        }
        printf("\n");
      }
#endif

  for (unsigned int y = tidy; y < ylimit; y += ystride) {
    for (unsigned int x = tidx; x < xlimit; x += xstride) {

      if (tidy == 0 && tidx == 0) {
        // Load from global into shared memory
        for (int yy = 0; yy < YBUFFER; yy++) {
          for (int xx = 0; xx < XBUFFER; xx++) {
            // printf("(%d, %d) %d\n", xx, yy, ((y+yy) * xlen) + (x + xx));
            buffer[yy][xx] = in[((y+yy) * xlen) + (x + xx)];
          }
        }
      }
      __syncthreads();

#ifdef DEBUG
/*
      printf("Buffer value\n");
      for (int yy = 0; yy < YBUFFER; yy++) {
        for (int xx = 0; xx < XBUFFER; xx++) {
          printf("%f ", buffer[yy][xx]);
        }
        printf("\n");
      }
*/
#endif

      // printf("x: %d y: %d\n", x, y);
      int pos[NDIM];
      int blc[NDIM];
      int trc[NDIM];

      pos[0] = tidx;
      pos[1] = tidy;

      blc[0] = pos[0];
      blc[1] = pos[1];

      trc[0] = blc[0] + BXLEN;
      trc[1] = blc[1] + BYLEN;

#ifdef DEBUG
/*
      printf("Shared Mem (%d %d)\n", tidx, tidy);
      for (int yy = blc[1]; yy < trc[1]; yy++) {
        for (int xx = blc[0]; xx < trc[0]; xx++) {
          printf("%f ", buffer[yy][xx]);
        }
        printf("\n");
      }

      printf("Global (%d %d)\n", tidx, tidy);
      for (int yy = y; yy < y + BYLEN; yy++) {
        for (int xx = x; xx < x + BXLEN; xx++) {
          printf("%f ", in[yy * xlen + xx]);
        }
        printf("\n");
      }
*/
      printf("Mem Box: (%d, %d) (%d, %d)\n", blc[0], blc[1], trc[0], trc[1]);
      printf("Actual Box: (%d, %d) (%d, %d)\n", x, y, x + BXLEN, y + BYLEN);

      for (int ya = y, ys = blc[1]; ya < y + BYLEN; ya++, ys++) {
        for (int xa = x, xs = blc[0]; xa < x + BXLEN; xa++, xs++) {
          float shm = buffer[ys][xs];
          float global = in[ya * xlen + xa];
          if (fabs(global - shm) > 0.001) {
            printf("Mismatch of values expected: %f actual: %f (%d, %d)\n",
                   global, shm, tidx, tidy);
          } else {
            printf("No problem (%d, %d)\n", tidx, tidy);
          }
        }
      }

#endif

      float sum = 0.0;
      // #pragma unroll 4
      for (int yy = blc[1]; yy < trc[1]; yy++) {
        for (int xx = blc[0]; xx < trc[0]; xx++) {
          int idx = yy * XBUFFER + xx;
          // Read from buffer rather than global mem
          sum += buffer[yy][xx];
        }
      }

      out[y * xlimit + x] = sum;
      __syncthreads();
    }
  }
}

__global__ void
without_mem_kernel(const float *in, float *out, const unsigned int xlen,
                   const unsigned int ylen, const unsigned int xlimit,
                   const unsigned int ylimit) {
  unsigned int ystride = blockDim.y * gridDim.y;
  unsigned int xstride = blockDim.x * gridDim.x;

  // printf("xstride: %d ystride: %d\n", xstride, ystride);
  // printf("x: %d y: %d\n", blockIdx.x * blockDim.x + threadIdx.x,
  //        blockIdx.y * blockDim.y + threadIdx.y);

  for (unsigned int y = blockIdx.y * blockDim.y + threadIdx.y; y < ylimit;
       y += ystride) {
    for (unsigned int x = blockIdx.x * blockDim.x + threadIdx.x; x < xlimit;
         x += xstride) {
      // printf("x: %d y: %d\n", x, y);
      int pos[NDIM];
      int blc[NDIM];
      int trc[NDIM];

      pos[0] = x;
      pos[1] = y;

      blc[0] = pos[0];
      blc[1] = pos[1];

      trc[0] = blc[0] + BXLEN;
      trc[1] = blc[1] + BYLEN;

      // printf("Box: (%d, %d) (%d, %d)\n", blc[0], blc[1], trc[0], trc[1]);

      float sum = 0.0;
      // #pragma unroll
      for (int yy = blc[1]; yy < trc[1]; yy++) {
        for (int xx = blc[0]; xx < trc[0]; xx++) {
          int idx = yy * xlen + xx;
          sum += in[idx];
        }
      }

      out[y * xlimit + x] = sum;
    }
  }
}

void host_reference(const float *in, float *out, const unsigned int xlen,
                    const unsigned int ylen, const unsigned int xlimit,
                    const unsigned int ylimit) {

#pragma omp parallel for collapse(2)
  for (int y = 0; y < ylimit; y++) {
    for (int x = 0; x < xlimit; x++) {
      int pos[NDIM];
      int blc[NDIM];
      int trc[NDIM];

      pos[0] = x;
      pos[1] = y;

      blc[0] = pos[0];
      blc[1] = pos[1];

      trc[0] = blc[0] + BXLEN;
      trc[1] = blc[1] + BYLEN;

      float sum = 0.0;

      for (int yy = blc[1]; yy < trc[1]; yy++) {
        for (int xx = blc[0]; xx < trc[0]; xx++) {
          int idx = yy * xlen + xx;
          sum += in[idx];
        }
      }

      out[y * xlimit + x] = sum;
    }
  }
}

void gpu_run_kernel(
    const float *arr, float *res, int shape[NDIM], int resShape[NDIM],
    int hboxsz[NDIM],
    void kernel(const float *in, float *out, const unsigned int xlen,
                const unsigned int ylen, const unsigned int xlimit,
                const unsigned int ylimit)) {
  size_t arrSize = sizeof(float) * shape[0] * shape[1];
  size_t resSize = sizeof(float) * resShape[0] * resShape[1];

  float *d_arr;
  float *d_res;

  hipEvent_t start, stop;

  checkCuda(hipEventCreate(&start));
  checkCuda(hipEventCreate(&stop));
  // printf("Starting\n");

  checkCuda(hipEventRecord(start));
  // printf("Starting 2\n");

  checkCuda(hipMalloc(&d_res, resSize));
  checkCuda(hipMalloc(&d_arr, arrSize));

  checkCuda(hipMemcpyAsync(d_arr, arr, arrSize, hipMemcpyHostToDevice));

// int grids = (N + BLOCKSIZE - 1) / BLOCKSIZE;
#ifndef GDEBUG
  dim3 blocks(BLOCKX, BLOCKY, 1);
  dim3 grids(GRIDX, GRIDY, 1);
#else
  dim3 blocks(2, 2, 1);
  dim3 grids(1, 1, 1);
#endif

  printf("blocksize: (%d, %d, %d) grid: (%d, %d, %d)\n", blocks.x, blocks.y,
         blocks.z, grids.x, grids.y, grids.z);

  // int shared_mem =
  //     (grids.x + hboxsz[0] - 1) * (grids.y + hboxsz[1] - 1) * sizeof(float) *
  //     2;
  int shared_mem = 16 * 1024;
  printf("Shared memory usage: %lu\n", shared_mem);
  // Run the kernel
  kernel<<<grids, blocks, shared_mem>>>(d_arr, d_res, shape[0], shape[1],
                                        resShape[0], resShape[1]);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("Error execting kernel %s\n", hipGetErrorString(error));
    assert(error == hipSuccess);
  }

  // printf("Finished kernel\n");

  // cudaDeviceSynchronize();

  // printf("Synched\n");

  checkCuda(hipMemcpyAsync(res, d_res, resSize, hipMemcpyDeviceToHost));

  // printf("Copy back\n");

  checkCuda(hipEventRecord(stop));
  checkCuda(hipEventSynchronize(stop));

  float elapsedTime;
  checkCuda(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("Elapsed GPU time %f ms\n", elapsedTime);

  checkCuda(hipFree(d_arr));
  checkCuda(hipFree(d_res));

  checkCuda(hipEventDestroy(start));
  checkCuda(hipEventDestroy(stop));
}

void experiment(int shape[NDIM], int hboxsz[NDIM], int mem) {
  int resShape[NDIM] = {shape[0] - hboxsz[0], shape[1] - hboxsz[1]};

  size_t arrSize = sizeof(float) * shape[0] * shape[1];
  size_t resSize = sizeof(float) * resShape[0] * resShape[1];

  float *arr = (float *)malloc(arrSize);

  double a = 5.0;

  for (int y = 0; y < shape[1]; y++) {
    for (int x = 0; x < shape[0]; x++) {
      float val = (double)std::rand() / (double)(RAND_MAX / a);
      arr[y * shape[0] + x] = y * shape[0] + x;
      // arr[y * shape[0] + x] = val;
    }
  }

#ifdef DEBUG
/*
  printf("Data\n");
  printf("------------------------------\n");
  printf("\nArray\n");
  for (int y = 0; y < shape[1]; y++) {
    for (int x = 0; x < shape[0]; x++) {
      float val = arr[y * shape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }
*/
#endif

  // size_t shared_mem = mem * 1024 * 1024;
  // printf("Setting Malloc Heap size %lu\n", shared_mem);
  // cudaDeviceSetLimit(cudaLimitMallocHeapSize, shared_mem);

  float *res_cpu = (float *)malloc(sizeof(float) * resShape[0] * resShape[1]);
  float *res_gpu_without_smem =
      (float *)malloc(sizeof(float) * resShape[0] * resShape[1]);
  float *res_gpu = (float *)malloc(sizeof(float) * resShape[0] * resShape[1]);

  printf("Array size %d %d\n", shape[0], shape[1]);
  printf("Window size %d %d\n", hboxsz[0], hboxsz[1]);

  auto t1 = std::chrono::high_resolution_clock::now();

  host_reference(arr, res_cpu, shape[0], shape[1], resShape[0], resShape[1]);

  auto t2 = std::chrono::high_resolution_clock::now();
  printf("Time taken CPU Grid: %ld ms\n",
         duration_cast<std::chrono::milliseconds>(t2 - t1).count());

  // cudaDeviceSetLimit(cudaLimitMallocHeapSize, 32 * 1024 * 1024);

  printf("Running shared memory test\n");
  gpu_run_kernel(arr, res_gpu, shape, resShape, hboxsz, shared_mem_kernel);

  printf("Running without shared memory test\n");
  gpu_run_kernel(arr, res_gpu_without_smem, shape, resShape, hboxsz,
                 without_mem_kernel);

#ifdef DEBUG
  printf("Result 1\n");
  printf("------------------------------\n");
  printf("\nExpected\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float val = res_cpu[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }

  printf("\nActual\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float val = res_gpu[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }

  printf("\nActual without shared mem\n");
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float val = res_gpu_without_smem[y * resShape[0] + x];
      printf("%f ", val);
    }
    printf("\n");
  }
#endif

#ifdef ASSERT
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float actual = res_gpu[y * resShape[0] + x];
      float expected = res_cpu[y * resShape[0] + x];
      if (fabs(actual - expected) > TOLERANCE) {
        fprintf(stderr,
                "Assertion failed value at %d %d expected: %f actual: %f\n", x,
                y, expected, actual);
      }
    }
  }
  printf("Assertions complete kernel\n");
  /*
  for (int y = 0; y < resShape[1]; y++) {
    for (int x = 0; x < resShape[0]; x++) {
      float actual = res_gpu_without_smem[y * resShape[0] + x];
      float expected = res_cpu[y * resShape[0] + x];
      if (fabs(actual - expected) > TOLERANCE) {
        fprintf(stderr,
                "Assertion failed value at %d %d expected: %f actual: %f\n",
                x, y, expected, actual);
      }
    }
  }
  */
  printf("Assertions complete for kernel with shared mem\n");
#endif

  free(res_gpu_without_smem);
  free(res_gpu);
  free(res_cpu);

  free(arr);
}

int main(int argc, char **argv) {
  if (argc < 2) {
    fprintf(stderr, "Usage:\n exp-9 <SIZE>\n");
    return 1;
  }
  int dim = std::atoi(argv[1]);
  int shape[NDIM] = {dim, dim};
  int hboxsz[NDIM] = {BXLEN, BYLEN};
  // int mem = std::atoi(argv[2]);
  int mem = 0; // this is not used

  experiment(shape, hboxsz, mem);
}
