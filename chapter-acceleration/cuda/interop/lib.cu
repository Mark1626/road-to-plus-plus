// #include "lib.cuh"

#include <hip/hip_runtime.h>
#include <cstddef>
#include <cstdio>

__global__ void saxpy_kernel(int N, float *a, float *b, float *c) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  if (index < N) {
    c[index] = a[index] * 5.0 + b[index];
  }
}

__global__ void hello() {
  printf("Hello World\n");
}

void saxpy(int N, float *a, float* b, float *c) {
  float* d_a;
  float* d_b;
  float* d_c;
  size_t arrSize = N * sizeof(float);

  hipMalloc(&d_a, arrSize);
  hipMalloc(&d_b, arrSize);
  hipMalloc(&d_c, arrSize);

  hipMemcpy(d_a, a, arrSize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, arrSize, hipMemcpyHostToDevice);

  int threads = 128;
  int blocksPerGrid = (N + threads - 1) / threads;

  saxpy_kernel <<< blocksPerGrid, threads >>> (N, d_a, d_b, d_c);

  hipDeviceSynchronize();

  hipMemcpy(c, d_c, arrSize, hipMemcpyDeviceToHost);

  hipFree(d_c);
  hipFree(d_b);
  hipFree(d_a);
}
